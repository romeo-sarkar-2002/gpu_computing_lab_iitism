#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 1024
__global__ void calcSqOfDiff (double *a, double *b, double *c)
{
    int i = blockIdx.x;
    if (i < N)
    {
        c[i] = a[i] - b[i];
        c[i] *= c[i];
    }
    return;
}

__global__ void calcSq (double *a, double *b)
{
    int i = blockIdx.x;
    if (i < N)
    {
        b[i] = a[i] * a[i];
    }
    return;
}

int main (int argc, char **argv)
{
    double x[N], y[N], z[N];
    double *dev_x, *dev_y, *dev_z;
    // allocate memory on device
    hipMalloc ((void **) (&dev_x), N * sizeof (double));
    hipMalloc ((void **) (&dev_y), N * sizeof (double));
    hipMalloc ((void **) (&dev_z), N * sizeof (double));
    for (int c = 0, i = 1; c < N; c++, i++)
    {
        x[c] = i * i;
        y[c] = 2 * i + 1;
    }

    // Copy data from host to device
    hipMemcpy (dev_x, x, N * sizeof (double), hipMemcpyHostToDevice);
    hipMemcpy (dev_y, y, N * sizeof (double), hipMemcpyHostToDevice);

    // launch kernel
    calcSqOfDiff <<<N, 1>>> (dev_x, dev_y, dev_z);
    hipDeviceSynchronize ();
    // wait for kernel to return
    // Copy result from device to host
    hipMemcpy (z, dev_z, N * sizeof (double), hipMemcpyDeviceToHost);
    double sumOfSq = 0;
    for (int c = 0; c < N; c++)
    {
        // printf ("%lf\n", z[c]);
        sumOfSq += z[c];
    }
    // printf ("sum of squares: %lf\n", sumOfSq);
    printf ("Distance between x and y is %lf\n", sqrt (sumOfSq));
    // calculating norm of x
    calcSq <<<N, 1>>> (dev_x, dev_z); 
    hipDeviceSynchronize ();
    hipMemcpy (z, dev_z, N * sizeof (double), hipMemcpyDeviceToHost);
    double sumOfSq_x = 0;
    for (int c = 0; c < N; c++)
    {
        // printf ("%lf\n", z[c]);
        sumOfSq_x += z[c];
    }
    printf ("Norm of x: %lf\n", sqrt (sumOfSq_x));
    // calculating norm of y
    calcSq <<<N, 1>>> (dev_y, dev_z); 
    hipDeviceSynchronize ();
    hipMemcpy (z, dev_z, N * sizeof (double), hipMemcpyDeviceToHost);
    double sumOfSq_y = 0;
    for (int c = 0; c < N; c++)
    {
        // printf ("%lf\n", z[c]);
        sumOfSq_y += z[c];
    }
    printf ("Norm of y: %lf\n", sqrt (sumOfSq_y));
    hipFree (dev_x);
    hipFree (dev_y);
    hipFree (dev_z);
    hipDeviceReset ();
    return 0;
}