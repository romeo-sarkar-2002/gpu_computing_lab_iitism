#include <hip/hip_runtime.h>

#include <stdio.h>
__global__ void increment (int *a)
{
    // printf ("*a: %d ", *a);
    (*a) = (*a) + 10;
    
    // printf ("*a: %d ", *a);
    return;
}

int main ()
{
    int a = 0;
    int *dev;
    hipMalloc (&dev, sizeof (int));
    hipMemset (dev, 0, sizeof (int));
    increment <<<1, 1024>>> (dev);
    hipDeviceSynchronize ();
    hipMemcpy (&a, dev, sizeof (int), hipMemcpyDeviceToHost);
    printf ("%d\n", a);
    return 0;
}