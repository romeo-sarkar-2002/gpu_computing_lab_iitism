#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10
__global__ void VecAddGPU (int *a, int *b, int *c)
{
    int i = blockIdx.x;
    if (i < N)
    {
        c[i] = a[i] + b[i];
    }
    return;
}

int main (int argc, char **argv)
{
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    // allocate memory on device
    hipMalloc ((void **) (&dev_a), N * sizeof (int));
    hipMalloc ((void **) (&dev_b), N * sizeof (int));
    hipMalloc ((void **) (&dev_c), N * sizeof (int));
    for (int i = 0; i < N; i++)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    // Copy data from host to device
    hipMemcpy (dev_a, a, N * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy (dev_b, b, N * sizeof (int), hipMemcpyHostToDevice);

    // launch kernel
    VecAddGPU <<<N, 1>>> (dev_a, dev_b, dev_c);
    // Copy result from device to host
    hipMemcpy (c, dev_c, N * sizeof (int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; i++)
    {
        printf ("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_c);
    return 0;
}