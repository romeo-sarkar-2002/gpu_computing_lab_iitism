#include<hip/hip_runtime.h>
#include<stdio.h>
#include "display.cuh"
#include "kernel.cuh"
#include "init.cuh"
#define N 8
#define TILE_WIDTH 2

int main(){
	//set up data size of matrix
	int Width = N;
	int nx = Width;
	int ny = Width;
	int nxy = nx * ny;
	int nBytes = nxy * sizeof(float);
	printf("Matrix size: %d by %d\n", nx, ny);
	printf("Tile size: %d by %d\n", TILE_WIDTH, TILE_WIDTH);
	
	//Malloc host memory
	float *h_A,*h_B,*h_C;
	h_A = (float *)malloc(nBytes);
	h_B = (float *)malloc(nBytes);
	h_C = (float *)malloc(nBytes);

	//initialize data at host side
	initialData(h_A,nxy);
	initialData(h_B,nxy);
	
	//Malloc device global memory
	float *d_MatA, *d_MatB, *d_MatC;
	hipMalloc((void **)&d_MatA, nBytes);
	hipMalloc((void **)&d_MatB, nBytes);
	hipMalloc((void **)&d_MatC, nBytes);

	//transfer data from host to device
	hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);

	//invoke kernel at host side
	int bdimx = TILE_WIDTH;
	int bdimy = TILE_WIDTH;
	dim3 block(bdimx, bdimy);
	dim3 grid((nx + block.x - 1)/block.x, (ny + block.y - 1)/block.y);
	
	MatrixMulKernel<<<grid, block>>>(d_MatA, d_MatB, d_MatC, Width);
	hipDeviceSynchronize();
	
	// copy kernel result back to host side
	hipMemcpy(h_C, d_MatC, nBytes, hipMemcpyDeviceToHost);
	printf("Matrix A is=\n");
	displayMatrix(h_A, nx, ny);
	printf("Matrix B is=\n");
	displayMatrix(h_B, nx, ny);
	printf("The product of Matrix A and Matrix B is=\n");
	displayMatrix(h_C, nx, ny);
	
	//free device global memory
	hipFree(d_MatA);
	hipFree(d_MatB);
	hipFree(d_MatC);
	
	//free host memory
	free(h_A);
	free(h_B);
	free(h_C);
	
	//reset device
	hipDeviceReset();
	
	return 0;
}