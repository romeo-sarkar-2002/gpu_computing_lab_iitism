#include "hip/hip_runtime.h"
#include "DistKernel.h"
#include <stdio.h>

#define TPB 16
__device__ float distance (float x1, float x2)
{
    return sqrt ((x2 - x1) * (x2 - x1));
}
__global__ void distanceKernel (float *d_out, float *d_in, float ref)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const float x = d_in[i];
    d_out[i] = distance (x, ref);
    printf ("i = %2d: distance from %f to %f is %f.\n", i, ref, x, d_out[i]);
    return;
}
void distanceArray (float *out, float *in, float ref, int len)
{
    // declare pointers to device arrays
    float *d_in = 0;
    float *d_out = 0;
    // allocate memory for device arrays
    hipMalloc (&d_in, len * sizeof (float));
    hipMalloc (&d_out, len * sizeof (float));
    // copy input data from host to device
    hipMemcpy (d_in, in, len * sizeof (float), hipMemcpyHostToDevice);
    // launch kernel to compute and store distance values
    distanceKernel <<<len / TPB, TPB>>> (d_out, d_in, ref);
    hipDeviceSynchronize ();
    hipMemcpy (out, d_out, len * sizeof (float), hipMemcpyDeviceToHost);
    // free the memory allocated for device arrays
    hipFree (d_in);
    hipFree (d_out);


}
