
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void print_name ()
{
    for (int i = 0; i < 10; i++)
        printf ("GPU> Romeo Sarkar (%d)\n", i + 1);
    return;
}
int main ()
{
    for (int i = 0; i < 10; i++)
        printf ("CPU> Romeo Sarkar (%d)\n", i + 1);
    print_name <<<1, 1>>> ();
    hipDeviceReset ();
    return 0;
}