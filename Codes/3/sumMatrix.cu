#include <hip/hip_runtime.h>
#include <stdio.h>

void initialData (float *ip, const int size)
{
    int i;
    for (i = 0; i < size; i++)
    {
        ip[i] = i;
    }
    return;
}

void displayMatrix (float *A, int nx, int ny)
{
    int idx;
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            idx = i * ny + j;
            printf ("%6.2f ", A[idx]);
        }
        printf ("\n");
    }
}

__global__ void sumMatrixOnGPU (float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < nx)
    {
        for (int iy = 0; iy < ny; iy++)
        {
            int idx = iy * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}

int main ()
{
    int nx = 4;
    int ny = 5;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof (float);
    // malloc host memory
    float *h_A, *h_B, *h_C;
    h_A = (float *) malloc (nBytes);
    h_B = (float *) malloc (nBytes);
    h_C = (float *) malloc (nBytes);
    //
    initialData (h_A, nxy);
    initialData (h_B, nxy);

    float *d_MatA, *d_MatB, *d_MatC;
    hipMalloc (&d_MatA, nBytes);
    hipMalloc (&d_MatB, nBytes);
    hipMalloc (&d_MatC, nBytes);

    hipMemcpy (d_MatA, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy (d_MatB, h_B, nBytes, hipMemcpyHostToDevice);


    int dimx  = 32;
    dim3 block (dimx, 1);
    dim3 grid ((nx + block.x - 1) / block.x, 1);

    sumMatrixOnGPU <<<grid, block>>> (d_MatA, d_MatB, d_MatC, nx, ny);

    hipDeviceSynchronize ();
    hipMemcpy (h_C, d_MatC, nBytes, hipMemcpyDeviceToHost);
    displayMatrix (h_C, nx, ny);

    hipFree (d_MatA);
    hipFree (d_MatB);
    hipFree (d_MatC);

    free (h_A);
    free (h_B);
    free (h_C);

    hipDeviceReset ();
    return (0);
}
