#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>


#define PRECISION_FIELD 0
#define SHOW_FUNCTION_CALLS 1
#define WARNINGS 0

__global__ void initialize_GPU (double *p, int rows, int cols);
__global__ void addKernel (double *m1, double *m2, double *a, int rows, int cols);
__global__ void subKernel (double *m1, double *m2, double *a, int rows, int cols);
__global__ void mulKernel (double *m1, double *m2, double *p, int rows, int x, int cols);
__global__ void transposeKernel (double *m1, double *m2, int rows, int cols);

void initialize (double *p, int rows, int cols);

struct Matrix
{
    int rows, cols;
    double *device_pointer, *host_pointer;

    Matrix () : rows (0), cols (0), device_pointer (NULL), host_pointer (NULL)
    {
        return;
    }
    Matrix (int r, int c) : Matrix ()
    {
        rows = r;
        cols = c;
        alloc ();
        return;
    }
    Matrix (const Matrix &M)
    {
        #if SHOW_FUNCTION_CALLS == 1
        printf ("\033[90mMatrix (const Matrix &M)\033[m\n");
        #endif
        rows = M.rows;
        cols = M.cols;
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        hipMemcpy (device_pointer, M.device_pointer, rows * cols * sizeof (double), hipMemcpyDeviceToDevice);
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        memcpy (host_pointer, M.host_pointer, rows * cols * sizeof (double));
        return;
    }
    Matrix (Matrix &&M)
    {
        #if SHOW_FUNCTION_CALLS == 1
        printf ("\033[90mMatrix (Matrix &&M)\033[m\n");
        #endif
        rows = M.rows;
        cols = M.cols;
        device_pointer = M.device_pointer;
        host_pointer = M.host_pointer;
        M.rows = M.cols = 0;
        M.device_pointer = M.host_pointer = NULL;
        return;
    }
    Matrix operator = (Matrix &M)
    {
        #if SHOW_FUNCTION_CALLS == 1
        printf ("\033[90mMatrix operator = (Matrix &M)\033[m\n");
        #endif
        clear ();
        rows = M.rows;
        cols = M.cols;
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        hipMemcpy (device_pointer, M.device_pointer, rows * cols * sizeof (double), hipMemcpyDeviceToDevice);
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        memcpy (host_pointer, M.host_pointer, rows * cols * sizeof (double));
        return *this;
    }
    Matrix operator = (Matrix &&M)
    {
        #if SHOW_FUNCTION_CALLS == 1
        printf ("\033[90mMatrix operator = (Matrix &&M)\033[m\n");
        #endif
        rows = M.rows;
        cols = M.cols;
        device_pointer = M.device_pointer;
        host_pointer = M.host_pointer;
        M.rows = M.cols = 0;
        M.device_pointer = M.host_pointer = NULL;
        return *this;
    }
    ~Matrix ()
    {
        #if SHOW_FUNCTION_CALLS == 1
        printf ("\033[90m~Matrix () : %p, %p\033[m\n", device_pointer, host_pointer);
        #endif
        clear ();
        return;
    }
    void alloc ()
    {
        host_pointer = (double *) (malloc (rows * cols * sizeof (double)));
        hipMalloc (&device_pointer, rows * cols * sizeof (double));
        return;
    }
    void clear ()
    {
        if (NULL != device_pointer)
        {
            hipFree (device_pointer);
        }
        if (NULL != host_pointer)
        {
            free (host_pointer);
        }
        rows = cols = 0;
        device_pointer = host_pointer = NULL;
        return;
    }
    void display ()
    {
        if (NULL == host_pointer)
        {
            #if WARNINGS == 1
            printf ("\nIn function \'\e[33mprint_matrix_yu\e[m\':\n\e[35mwarning:\e[m \'m\' is (null)\n");
            #endif
            return;
        }
        #define BUFFER_SIZE 128
        int *max_width_arr = (int *) (malloc (cols * sizeof (int)));
        char **mat_of_strs = (char **) malloc (rows * cols * sizeof (char *));
        char *str;
        int width;
        for (size_t i = 0; i < cols; i++)
        {
            max_width_arr[i] = 1;
            for (size_t j = 0; j < rows; j++)
            {
                str = (char *) malloc (BUFFER_SIZE * sizeof (char));
                width = snprintf (str, BUFFER_SIZE, "%.*lf", PRECISION_FIELD, host_pointer[j * cols + i]);
                str = (char *) realloc (str, ((size_t) (width + 1)) * sizeof (char));
                mat_of_strs[j * cols + i] = str;
                if (max_width_arr[i] < width)
                    max_width_arr[i] = width;
            }
        }
        for (size_t i = 0; i < rows; i++)
        {
            printf ("\033[1;32m\xb3\033[m");
            for (size_t j = 0; j < cols; j++)
            {
                width = strlen (mat_of_strs[i * cols + j]);
                for (int x = 0; x < max_width_arr[j] - width; x++)
                    printf (" ");
                printf ("%s", mat_of_strs[i * cols + j]);
                if (j != (cols - 1))
                    printf (" ");
            }
            printf ("\033[1;32m\xb3\033[m");
            // newline:
            printf ("\n");
        }
        for (size_t i = 0; i < rows; i++)
            for (size_t j = 0; j < cols; j++)
                free (mat_of_strs[i * cols + j]);
        free (mat_of_strs);
        free (max_width_arr);
        return;
    }
    void initialize ()
    {
        ::initialize (host_pointer, rows, cols);
        H2D ();
        return;
    }
    void H2D ()
    {
        hipMemcpy (device_pointer, host_pointer, cols * rows * sizeof (double), hipMemcpyHostToDevice);
        return;
    }
    void D2H ()
    {
        hipMemcpy (host_pointer, device_pointer, cols * rows * sizeof (double), hipMemcpyDeviceToHost);
        return;
    }
    Matrix operator + (const Matrix &M)
    {
        if (rows != M.rows && cols != M.cols)
        {
            printf ("Matrix1 (%dX%d); Matrix2 (%dX%d)\n", rows, cols, M.rows, M.cols);
            return Matrix ();
        }
        Matrix p (rows, M.cols);
        dim3 block (1, 1, 1);
        dim3 grid (rows, M.cols, 1);
        addKernel <<< block, grid>>> (device_pointer, M.device_pointer, p.device_pointer, rows, cols);
        hipDeviceSynchronize ();
        p.D2H ();
        // p.display ();
        return p;
    }
    Matrix operator - (const Matrix &M)
    {
        if (rows != M.rows && cols != M.cols)
        {
            printf ("Matrix1 (%dX%d); Matrix2 (%dX%d)\n", rows, cols, M.rows, M.cols);
            return Matrix ();
        }
        Matrix p (rows, M.cols);
        dim3 block (1, 1, 1);
        dim3 grid (rows, M.cols, 1);
        subKernel <<< block, grid>>> (device_pointer, M.device_pointer, p.device_pointer, rows, cols);
        hipDeviceSynchronize ();
        p.D2H ();
        // p.display ();
        return p;
    }
    Matrix operator * (const Matrix &M)
    {
        if (cols != M.rows)
        {
            printf ("Matrix1 (%dX%d); Matrix2 (%dX%d)\n", rows, cols, M.rows, M.cols);
            return Matrix ();
        }
        Matrix p (rows, M.cols);
        dim3 block (1, 1, 1);
        dim3 grid (rows, M.cols, 1);
        mulKernel <<< block, grid>>> (device_pointer, M.device_pointer, p.device_pointer, rows, cols, M.cols);
        hipDeviceSynchronize ();
        p.D2H ();
        // p.display ();
        return p;
    }
    Matrix operator ~ ()
    {
        Matrix t (cols, rows);
        dim3 block (1, 1, 1);
        dim3 grid (rows, cols, 1);
        transposeKernel <<<grid, block>>> (device_pointer, t.device_pointer, rows, cols);
        hipDeviceSynchronize ();
        t.D2H ();
        return t;
    }
};

void initialize (double *p, int rows, int cols)
{
    for (int i = 0; i < rows * cols; i++)
    {
        p[i] = rand () % 21 - 10;
    }
    return;
}

__global__ void addKernel (double *m1, double *m2, double *a, int rows, int cols)
{
    int Row = blockIdx.x * blockDim.x + threadIdx.x;
    int Col = blockIdx.y * blockDim.y + threadIdx.y;
    if (Row < rows && Col < cols)
    {
        a[Row * cols + Col] = m1[Row * cols + Col] + m2[Row * cols + Col];
    }
    return;
}

__global__ void subKernel (double *m1, double *m2, double *a, int _rows, int _cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < _rows && col < _cols)
    {
        a[row * _cols + col] = m1[row * _cols + col] - m2[row * _cols + col];
    }
    return;
}

__global__ void mulKernel (double *m1, double *m2, double *p, int _rows, int _x, int _cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < _rows && col < _cols)
    {
        double a = 0;
        for (int k = 0; k < _x; k++)
        {
            a += m1[row * _x + k] * m2[k * _cols + col];
        }
        p[row * _cols + col] = a;
    }
    return;
}
__global__ void transposeKernel (double *m1, double *m2, int _rows, int _cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < _rows && col < _cols)
    {
        m2[col * _rows + row] = m1[row * _cols + col];
    }
    return;
}

#define DISPLAY(x) \
printf ("\033[4;31mMatrix " #x ":\033[m\n");\
x.display ();

int main ()
{
    srand (time (NULL));
    Matrix A (4, 4), B (4, 4);
    
    A.initialize (), B.initialize ();


    Matrix TA = ~A, TB = ~B;
    Matrix PAB = A * B;
    Matrix PTATB = TA * TB;
    Matrix D = PAB - PTATB;

    DISPLAY (A);
    DISPLAY (B);
    DISPLAY (TA);
    DISPLAY (TB);
    DISPLAY (PAB);
    DISPLAY (PTATB);
    DISPLAY (D);
    
    hipDeviceReset ();
    return 0;
}