#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
// macros
#define _SHARED_ARR_LEN_ 347U

#define ceil_div(a, b) (((a) + (b) - 1) / (b))
#define floor_div(a, b) ((a) / (b))
#include "Error.cuh"

__global__ void reduced_sum1 (double *arr, double *sum, size_t size)
{
    __shared__ double s_arr[_SHARED_ARR_LEN_];
    // unsigned int s = ceil_div (_SHARED_ARR_LEN_, 2);
    // unsigned int globalIdx = 2 * threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_;
    // if (2 * threadIdx.x < )
    unsigned int s;
    if (floor_div (size, 2 * _SHARED_ARR_LEN_) == blockIdx.x)
    {
        s = ceil_div (size % (2 * _SHARED_ARR_LEN_), 2);
    }
    else
    {
        s = _SHARED_ARR_LEN_;
    }
    if ((threadIdx.x < s) && ((2 * (threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_)) < size))
    {
        s_arr[threadIdx.x] = arr[2 * (threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_)];
    }
    // else
    // {
    //     goto finish_line;
    // }
    __syncthreads ();
    if ((threadIdx.x < s) && ((2 * (threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_) + 1) < size))
    {
        s_arr[threadIdx.x] += arr[2 * (threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_) + 1];
    }
    // if (0 == blockIdx.x)
    // {
    //     // for (unsigned int i = 0; i < _SHARED_ARR_LEN_; i++)
    //     // {
    //     printf ("%.0lf ", s_arr[threadIdx.x]);
    //     // }
    //     __syncthreads ();
    //     if (0 == threadIdx.x)
    //         printf ("\n");
    // }
    __syncthreads ();
    
    // unsigned int 
    // now, find the sum of the entire block:
    for (unsigned int stride = 1; stride < s; stride <<= 1)
    {
        if ((threadIdx.x % (stride << 1)) == 0)
        {
            if ((threadIdx.x + stride) < s)
            {
                s_arr[threadIdx.x] += s_arr[threadIdx.x + stride];
            }
        }
        // else
        // {
        //     goto finish_line;
        // }
        __syncthreads ();
    }
    sum[blockIdx.x] = s_arr[0];
    // finish_line:
    // printf ("<<%u;%u>>\n", blockIdx.x, threadIdx.x);
    // blockIdx.x
    return;
}
__global__ void reduced_sum2 (double *arr, double *sum, unsigned int size)
{
    __shared__ double s_arr[_SHARED_ARR_LEN_];
    // #define s_arr arr

    unsigned int globalIdx = threadIdx.x + blockIdx.x * _SHARED_ARR_LEN_;
    if (globalIdx < size)
    {
        s_arr[threadIdx.x] = arr[globalIdx];
        // if (0 == blockIdx.x)
        // printf ("%.0lf ", arr[globalIdx]);
    }
    __syncthreads ();
    // __syncthreads ();
    // if (threadIdx.x == 0 && blockIdx.x == 0)
    //     printf ("\n\n\n\n");
    // adding the entire block
    unsigned int trailing_stride, stride;
    if (floor_div (size, _SHARED_ARR_LEN_) == blockIdx.x)
    {
        // printf ("\033[90mH\033[m");
        trailing_stride = size % _SHARED_ARR_LEN_;
    }
    else
    {
        // printf ("\033[90mX\033[m");
        trailing_stride = _SHARED_ARR_LEN_;
    }
    stride = ceil_div (trailing_stride, 2);
    for (; trailing_stride > 1; trailing_stride = stride, stride = ceil_div (stride, 2))
    {
        if (threadIdx.x < stride)
        {
            if ((threadIdx.x + stride) < trailing_stride)
            {
                s_arr[threadIdx.x] += s_arr[threadIdx.x + stride];
                // if (blockIdx.x == 0)
                // {
                //     printf ("\033[32m%d->%.0lf\033[m ", threadIdx.x, s_arr[threadIdx.x]);
                // }
            }
            // __syncthreads ();
        }
        else
        {
            goto finish_line;
        }
        // __syncthreads ();
        // if (blockIdx.x == 0 && threadIdx.x == 0)
        // {
        //     printf ("\n\n\n\n");
        // }
        __syncthreads ();
    }
    // if (0 == threadIdx.x)
    // {
    //     printf ("\033[32m%.1lf\033[m ", s_arr[0]);
    // }
    // if (0 == blockIdx.x && threadIdx.x == 0)
    // {
    //     printf ("\033[31m%lf\033[m\n", s_arr[0]);
    // }
    if (0 == threadIdx.x)
    {
        sum[blockIdx.x] = s_arr[0];
    }
    // sum[blockIdx.x] = arr[blockIdx.x * _SHARED_ARR_LEN_];
    finish_line:
    return;
}
double calculate_sum_cpu (double *arr, size_t size)
{
    double s = 0;
    for (size_t i = 0; i < size; i++)
    {
        s += arr[i];
    }
    return s;
}
double calculate_sum_cpu (double *arr, size_t startIdx, size_t endIdx)
{
    double s = 0;
    for (int i = startIdx; i < endIdx; i++)
    {
        s += arr[i];
    }
    return s;
}
void initialize_array (double *arr, size_t size)
{
    struct timespec start, stop;
    timespec_get (&start, TIME_UTC);
    for (size_t i = 0; i < size; i++)
    {
        // arr[i] = ((double) rand ()) * ((double) (rand ()));
        arr[i] = (double) rand ();
        // arr[i] = i + 1;
        // arr[i] = 1;
    }
    timespec_get (&stop, TIME_UTC);
    printf ("time taken to initialize the array: %.9lf secs.\n", ((double) (stop.tv_nsec - start.tv_nsec) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec))));
    return;
}
int cmp (const void *a, const void *b)
{
    const double *x = (const double *) (a), *y = (const double *) (b);
    if (x < y)
    {
        return 0; // i.e. don't swap
    }
    else
    {
        return 1; // i.e. swap
    }
}
void sort_array (double *arr, size_t size)
{
    struct timespec start, stop;
    timespec_get (&start, TIME_UTC);
    qsort (arr, size, sizeof (double), cmp);
    timespec_get (&stop, TIME_UTC);
    printf ("time taken to sort the array: %.9lf secs.\n", ((double) (stop.tv_nsec - start.tv_nsec)) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec)));
    return;
}
void print_array (double *arr, size_t size)
{
    for (size_t i = 0; i < size; i++)
    {
        printf ("%.0f ", arr[i]);
    }
    printf ("\n");
    return;
}
double calculate_sum_gpu1 (double *arr, size_t size)
{
    // double *dev_arr;
    // hipMalloc (&dev_arr, size);
    
    // hipMemcpy (dev_arr, arr, size * sizeof (double), hipMemcpyHostToDevice);
    // array will be divided into smaller array of size _SHARED_ARR_LEN_
    double sum;
    size_t temp_arr_size = size, temp_sum_arr_size = ceil_div (temp_arr_size, 2 * _SHARED_ARR_LEN_);
    double *dev_temp_arr = NULL, *dev_temp_sum_arr = NULL;
    hipMalloc (&dev_temp_arr, sizeof (double) * temp_arr_size);
    hipMemcpy (dev_temp_arr, arr, temp_arr_size * sizeof (double), hipMemcpyHostToDevice);
    for (; temp_arr_size > 1; temp_arr_size = temp_sum_arr_size, temp_sum_arr_size = ceil_div (temp_arr_size, 2 * _SHARED_ARR_LEN_))
    {
        // temp_size = ceildiv (temp_size, _SHARED_ARR_LEN_);
        // temp_arr = (double *) (malloc (sizeof (double) * temp_arr_size));
        // printf ("launch param: <<< %zu, %u >>>\n", temp_sum_arr_size, _SHARED_ARR_LEN_);
        // printf ("launch param1: %u\n", ceil_div (12, 5));
        chkError (hipMalloc (&dev_temp_sum_arr, sizeof (double) * temp_sum_arr_size))
        reduced_sum1 <<< temp_sum_arr_size, _SHARED_ARR_LEN_ >>> (dev_temp_arr, dev_temp_sum_arr, temp_arr_size);
        getLastError ();
        hipDeviceSynchronize ();
        // printf ("launch param2: %zu, %zu\n", temp_arr_size, temp_sum_arr_size);
        // printf ("\n");
        // comment:
        // double *p = (double *) malloc (temp_sum_arr_size * sizeof (double));
        // hipMemcpy (p, dev_temp_sum_arr, sizeof (double) * temp_sum_arr_size, hipMemcpyDeviceToHost);
        // double t;
        // for (int i = 0; i < temp_sum_arr_size; i++)
        // {
        //     printf ("\033[31m%.1lf\033[m ", p[i]);
        // }
        // printf ("\n");
        // for (int i = 0; i < temp_sum_arr_size; i++)
        // {
        //     if (p[i] != (t = calculate_sum_cpu (arr, i * 2 * _SHARED_ARR_LEN_, (i + 1) * 2 * _SHARED_ARR_LEN_)))
        //     {
        //         printf ("error: %d; %.1lf instead of %.1lf\n", i, p[i], t);
        //         exit (0);
        //     }
        // }
        // exit (0);
        // comment:

        hipFree (dev_temp_arr);
        dev_temp_arr = dev_temp_sum_arr;
        dev_temp_sum_arr = NULL;
        // return 0;
        
    }
    hipMemcpy (&sum, dev_temp_arr, sizeof (double), hipMemcpyDeviceToHost);
    hipFree (dev_temp_arr);
    return sum;
}
double calculate_sum_gpu2 (double *arr, size_t size)
{
    // double *dev_arr;
    // hipMalloc (&dev_arr, size);
    
    // hipMemcpy (dev_arr, arr, size * sizeof (double), hipMemcpyHostToDevice);
    // array will be divided into smaller array of size _SHARED_ARR_LEN_
    double sum;
    size_t temp_arr_size = size, temp_sum_arr_size = ceil_div (size, _SHARED_ARR_LEN_);
    double *dev_temp_arr = NULL, *dev_temp_sum_arr = NULL;
    hipMalloc (&dev_temp_arr, sizeof (double) * temp_arr_size);
    hipMemcpy (dev_temp_arr, arr, temp_arr_size * sizeof (double), hipMemcpyHostToDevice);
    for (; temp_arr_size > 1; temp_arr_size = temp_sum_arr_size, temp_sum_arr_size = ceil_div (temp_sum_arr_size, _SHARED_ARR_LEN_))
    {
        // temp_size = ceildiv (temp_size, _SHARED_ARR_LEN_);
        // temp_arr = (double *) (malloc (sizeof (double) * temp_arr_size));
        hipMalloc (&dev_temp_sum_arr, sizeof (double) * temp_sum_arr_size);
        reduced_sum2 <<< temp_sum_arr_size, _SHARED_ARR_LEN_ >>> (dev_temp_arr, dev_temp_sum_arr, temp_arr_size);
        getLastError ();
        hipDeviceSynchronize ();
        // printf ("\n");
        // comment:
        // double *p = (double *) malloc (temp_sum_arr_size * sizeof (double));
        // hipMemcpy (p, dev_temp_sum_arr, sizeof (double) * temp_sum_arr_size, hipMemcpyDeviceToHost);
        // double t;
        // for (int i = 0; i < temp_sum_arr_size && i < 1; i++)
        // {
        //     if (p[i] != (t = calculate_sum_cpu (arr, i * _SHARED_ARR_LEN_, (i + 1) * _SHARED_ARR_LEN_)))
        //     {
        //         printf ("error: %d; %.1lf instead of %.1lf\n", i, p[i], t);
        //     }
        // }
        // comment:
        /*
        f
        sf
        sd

        */

        hipFree (dev_temp_arr);
        dev_temp_arr = dev_temp_sum_arr;
        dev_temp_sum_arr = NULL;
        // return 0;
        
    }
    hipMemcpy (&sum, dev_temp_arr, sizeof (double), hipMemcpyDeviceToHost);
    hipFree (dev_temp_arr);
    return sum;
}
int sum (double *arr, size_t size)
{
    struct timespec start, stop;
    // timespec_get (&start, TIME_UTC);
    // clock_t st = clock ();
    sort_array (arr, size);
    // printf ("time: %.3lf secs.\n", ((double) (clock () - st)) / CLOCKS_PER_SEC);
    // timespec_get (&stop, TIME_UTC);
    // printf ("time taken to sort the array: %.9lf secs.\n", ((double) (stop.tv_nsec - start.tv_nsec)) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec)));
    /* = = = = = = = = = = = = = */
    timespec_get (&start, TIME_UTC);
    double sum_cpu = calculate_sum_cpu (arr, size);
    timespec_get (&stop, TIME_UTC);
    printf ("sum_cpu time: %.9lf secs.\n", ((double) (stop.tv_nsec - start.tv_nsec) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec))));
    /*= = = = = = = = = = = = = */
    timespec_get (&start, TIME_UTC);
    double sum_gpu1 = calculate_sum_gpu1 (arr, size);
    timespec_get (&stop, TIME_UTC);
    printf ("sum_gpu1 time: %.9lf secs.\n", ((double) (stop.tv_nsec - start.tv_nsec) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec))));
    /*= = = = = = = = = = = = = */
    timespec_get (&start, TIME_UTC);
    double sum_gpu2 = calculate_sum_gpu2 (arr, size);
    timespec_get (&stop, TIME_UTC);
    printf ("sum_gpu2 time: %.9lf secs. \033[90m(less warp divergence)\033[m\n", ((double) (stop.tv_nsec - start.tv_nsec) * 1e-9 + ((double) (stop.tv_sec - start.tv_sec))));
    /*= = = = = = = = = = = = = */
    printf ("{sum_cpu, sum_gpu1, sum_gpu2} = {%.0lf, %.0lf, %.0lf}\n", sum_cpu, sum_gpu1, sum_gpu2);
    if (sum_cpu != sum_gpu1)
    {
        printf ("\033[1;31merror\033[m: (sum_cpu != sum_gpu1)\n");
        return 1;
    }
    else
    {
        if (sum_cpu != sum_gpu2)
        {
            printf ("\033[1;31merror\033[m: (sum_cpu != sum_gpu2)\n");
            return 1;
        }
        else
        {
            return 0;
        }
    }
    // const int i = 0;
    // if (sum_cpu != sum_gpu2)
    // {
    //     printf ("\033[1;31merror\033[m: (sum_cpu != sum_gpu)\n");
    //     return 1;
    // }
    // else
    // {
    //     return 0;
    // }
    return 0;
}
double *allocate_array (size_t size)
{
    printf ("size of array: %zd Bytes (%.6lf GB)\n", sizeof (double) * size, ((double) (sizeof (double) * size)) / (1024.0 * 1024.0 * 1024.0));
    double *arr = (double *) (malloc (sizeof (double) * size));
    return arr;
}
int main ()
{
    srand (time (NULL));
    size_t size = 565786565; // array size;

    double *arr = allocate_array (size);
    initialize_array (arr, size);
    sum (arr, size);
    return 0;
}