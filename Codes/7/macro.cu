// #include <stdio.h>
#define last_error_check()\
    hipError_t err = hipGetLastError (); \
    if (err != hipSuccess || 1) \
    { \
        printf ("%s(%d): \033[1;31merror\033[m: %s i.e. %s\n", __FILE__, __LINE__, hipGetErrorString (err), hipGetErrorName (err));\
        exit err; \
    }
last_error_check ()