#include "hip/hip_runtime.h"
#include "lab9ex1lib.cuh"

int main()
{
	hipDeviceProp_t prop;
	int whichDevice;
	CHECK(hipGetDevice(&whichDevice));
	CHECK(hipGetDeviceProperties(&prop, whichDevice));
	if(!prop.deviceOverlap) {
		printf( "Device will not handle overlaps, so no speed up from streams\n");
		return 0;
	}
	
	hipStream_t stream0, stream1;
	CHECK(hipStreamCreate(&stream0));
	CHECK(hipStreamCreate(&stream1));

	rec_init;

	// a is array, b is sum, c is max
	int *host_a, *host_sum, *host_max;
	int *dev_a0, *dev_b0;
	int *dev_a1, *dev_b1;
	
	int bdimx = BD;
	int gdimx = (N + bdimx -1)/bdimx;
	dim3 block(bdimx);
	dim3 grid(gdimx);

	printf("Array Size is = %llu\n",N);
	// #pragma push_macro("CHECK")
	// #undef CHECK
	// #define CHECK(Call) Call
	CHECK(hipMalloc((void**)&dev_a0, N*sizeof(int)));
	CHECK(hipMalloc((void**)&dev_b0, gdimx*sizeof(int)));

	CHECK(hipMalloc((void**)&dev_a1, N*sizeof(int)));
	CHECK(hipMalloc((void**)&dev_b1, gdimx*sizeof(int)));

	CHECK(hipHostAlloc((void **)&host_a, N * sizeof(int), hipHostMallocDefault));
	CHECK(hipHostAlloc((void **)&host_sum, gdimx * sizeof(int), hipHostMallocDefault));
	CHECK(hipHostAlloc((void **)&host_max, gdimx * sizeof(int), hipHostMallocDefault));
	srand(time(0));
	initialize(host_a);
	show(host_a,N);

/*---------------------------------------------------------------------------------------------------------------------*/

	rec_start(stream0);
		CHECK(hipMemcpyAsync(dev_a0, host_a, N*sizeof(int),hipMemcpyHostToDevice, stream0));
		CHECK(hipStreamSynchronize(stream0));
	rec_stop(stream0);
	rec_pr("Time to do memory transfers from H2D in stream0:");

	rec_start(stream0);
		sumReduce<<<grid,block,0,stream0>>>(dev_a0,dev_b0);
		sumReduce<<<1,block,0,stream0>>>(dev_b0,dev_b0);
	rec_stop(stream0);
	rec_pr("TimeElapsed in computation in stream0:");

	rec_start(stream0);
		CHECK(hipMemcpyAsync(host_sum, dev_b0, sizeof(int),hipMemcpyDeviceToHost, stream0));
		CHECK(hipStreamSynchronize(stream0));
	rec_stop(stream0);
	rec_pr("Time to do memory transfers from D2H in stream0:");

	CHECK(hipStreamSynchronize(stream0));
	printf("Sum = %d\n",host_sum[0]);

/*---------------------------------------------------------------------------------------------------------------------*/

	rec_start(stream1);
		CHECK(hipMemcpyAsync(dev_a1, host_a, N*sizeof(int),hipMemcpyHostToDevice, stream1));
		CHECK(hipStreamSynchronize(stream1));
	rec_stop(stream1);
	rec_pr("Time to do memory transfers from H2D in stream1:");

	rec_start(stream1);
		arrMax<<<grid,block,0,stream1>>>(dev_a1,dev_b1);
		arrMax<<<1,block,0,stream1>>>(dev_b1,dev_b1);
	rec_stop(stream1);
	rec_pr("TimeElapsed in computation in stream1:");

	rec_start(stream1);
		CHECK(hipMemcpyAsync(host_max, dev_b1, sizeof(int),hipMemcpyDeviceToHost, stream1));
		CHECK(hipStreamSynchronize(stream1));
	rec_stop(stream1);
	rec_pr("Time to do memory transfers from D2H in stream0:");

	
	CHECK(hipStreamSynchronize(stream1));
	printf("Max = %d\n",host_max[0]);

/*---------------------------------------------------------------------------------------------------------------------*/
	// #pragma pop_macro("CHECK")

	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipHostFree(host_a);
	hipHostFree(host_sum);
	hipHostFree(host_max);
	CHECK(hipStreamDestroy(stream0));
	CHECK(hipStreamDestroy(stream1));
	hipDeviceReset();
	return 0;
}
