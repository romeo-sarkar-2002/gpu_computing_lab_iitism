#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define N (1024*1024)
#define FULL_DATA_SIZE (N*20)
#define CHECK(call)\
{\
    const hipError_t error = call;\
    if(error != hipSuccess)\
    {\
        fprintf (stderr,"Error:%s:%d,",__FILE__,__LINE__);\
        fprintf (stderr,"code:%d,reason:%s\n",error,\
        hipGetErrorString (error));\
        exit (1);\
    }\
}
__global__ void kernel (int  *a, int  *b, int  *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        c[idx] = (a[idx] + b[idx]) / 2.0;
    }
}

int main (void)
{
    hipDeviceProp_t prop;
    int whichDevice;
    CHECK (hipGetDevice (&whichDevice));
    CHECK (hipGetDeviceProperties (&prop, whichDevice));
    if (!prop.deviceOverlap)
    {
        printf ("Device will not handle overlaps, so no speed up from streams\n");
        return 0;
    }
    hipEvent_t start, stop;
    float elapsedTime;

    hipStream_t stream;
    int  *host_a, *host_b, *host_c;
    int  *dev_a, *dev_b, *dev_c;

    //start the timers
    CHECK (hipEventCreate (&start));
    CHECK (hipEventCreate (&stop));

    //initialize the stream
    CHECK (hipStreamCreate (&stream));
    //allocatethememoryontheGPU
    CHECK (hipMalloc ((void **) &dev_a, N * sizeof (int)));
    CHECK (hipMalloc ((void **) &dev_b, N * sizeof (int)));
    CHECK (hipMalloc ((void **) &dev_c, N * sizeof (int)));
        //allocate host locked memory, used to stream
    CHECK (hipHostAlloc ((void **) &host_a, FULL_DATA_SIZE * sizeof (int), hipHostMallocDefault));
    CHECK (hipHostAlloc ((void **) &host_b, FULL_DATA_SIZE * sizeof (int), hipHostMallocDefault));
    CHECK (hipHostAlloc ((void **) &host_c, FULL_DATA_SIZE * sizeof (int), hipHostMallocDefault));

    for (int i = 0;i < FULL_DATA_SIZE;i++)
    {
        host_a[i] = rand ();
        host_b[i] = rand ();
    }
    CHECK (hipEventRecord (start, 0));
    //now loop over full data,in bite−sized chunks
    for (int i = 0;i < FULL_DATA_SIZE;i += N)
    {
//copythelockedmemorytothedevice,async
        CHECK (hipMemcpyAsync (dev_a, host_a + i, N * sizeof (int), hipMemcpyHostToDevice, stream));
        CHECK (hipMemcpyAsync (dev_b, host_b + i, N * sizeof (int), hipMemcpyHostToDevice, stream));

        kernel <<<N / 256, 256, 0, stream >>> (dev_a, dev_b, dev_c);

        //copy the data fromdevicetolockedmemory
        CHECK (hipMemcpyAsync (host_c + i, dev_c,N * sizeof (int),hipMemcpyDeviceToHost, stream));

    }
    //copy result chunk from locked to full buffer
    CHECK (hipStreamSynchronize (stream));

    CHECK (hipEventRecord (stop, 0));

    CHECK (hipEventSynchronize (stop));
    CHECK (hipEventElapsedTime (&elapsedTime, start, stop));
    printf ("The single stream with ID %p was created and the total time taken for (data transfer, computation) is % 8.6f ms\n", stream, elapsedTime);
        //clean up the streams and memory
        CHECK (hipHostFree (host_a));
    CHECK (hipHostFree (host_b));
    CHECK (hipHostFree (host_c));
    CHECK (hipFree (dev_a));
    CHECK (hipFree (dev_b));
    CHECK (hipFree (dev_c));
    CHECK (hipStreamDestroy (stream));

    return 0;
}
